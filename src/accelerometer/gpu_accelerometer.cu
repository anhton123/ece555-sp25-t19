
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addArrays(int *a, int *b, int *c, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int size = 10;
    int bytes = size * sizeof(int);

    int h_a[10], h_b[10], h_c[10];
    for (int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, bytes);
    hipMalloc((void**)&d_b, bytes);
    hipMalloc((void**)&d_c, bytes);

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addArrays<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    printf("Result: ");
    for (int i = 0; i < size; i++) {
        printf("%d ", h_c[i]);
    }
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}

